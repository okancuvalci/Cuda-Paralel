#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define Rp 10001
#define Clm 99911
#define veri 99911
#define Width 10000
#define BLOCKSIZE 125


//Bloklara b�ld���m�z matris ve vekt�r�n �arp�m�//
__global__ void carpim(float *Md, float *Nd, float *Pd)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0;
	if (tid<Width){
		for (int i = 0; i<Width; i++)
			sum += Md[i + Width*tid] * Nd[i];
		Pd[tid] = sum;
	}
}
int main()
{
	clock_t start, end;
	start = clock();
	FILE *satir = fopen("Rp.txt", "r");
	FILE *sutun = fopen("C.txt", "r");
	FILE *data = fopen("D.txt", "r");
	int i = 0, s = 0, l = 0, z = 0;
	static int A[Rp], B[Rp - 1], C[Clm];
	static float D[veri];
	static float M[Width][Width], N[Width][1], sonuc[Width][1];
	//N matrisi olu�turuluyor//
	for (int i = 0; i < Width; i++)
	{
		N[i][0] = 1;
	}
	//Rp.txt dosyas�ndan okunan de�erler A matrisine at�l�yor//
	for (i = 0; i < Rp; i++)
	{
		fscanf(satir, "%d", &A[i]);
	}
	//A matrisindeki de�erlerin sonucu B matrisine aktar�l�yor//
	for (i = 0; i < Rp - 1; i++)
	{
		B[i] = A[i + 1] - A[i];
	}
	//C.txt dosyas�ndaki sutun bilgileri C matrisine at�l�yor//
	for (i = 0; i < Clm; i++)
	{
		fscanf(sutun, "%d", &C[i]);
	}
	//D.txt dosyas�ndaki veriler D matrisine aktar�l�yor//
	for (i = 0; i < veri; i++)
	{
		fscanf(data, "%f", &D[i]);

	}
	//M matrisinin t�m elemanlar�n� 0'a setliyoruz//
	for (int i = 0; i < Width; i++)
	{
		for (int j = 0; j < Width; j++)
		{
			M[i][j] = 0;
		}
	}
	//Dosyalardan okuyup s�rayla B C D matrislerine verdi�im de�erlere g�re//
	//i�lem yapacag�m 10000*10000'lik M matrisi olu�turuluyor//
	for (i = 0; i < Rp - 1; i++)
	{
		for (l = 0; l < B[i]; l++)
		{
			z = C[s];
			M[i][z] = D[s];
			s++;
		}
	}
	float *Md, *Nd, *Pd;
	float *d_Md, *d_Nd, *d_Pd;
	int a = 0, b = 0;
	//CPU da Md Nd ve Pd de�i�kenleri i�in alan tahsisi//
	Md = (float*)malloc(Width * Width * sizeof(float));
	Nd = (float*)malloc(Width * 1 * sizeof(float));
	Pd = (float*)malloc(Width * 1 * sizeof(float));
	for (int i = 0; i < Width; i++) {
		for (int j = 0; j < Width; j++) {
			Md[a] = M[i][j];
			a++;
		}
	}
	for (int i = 0; i < Width; i++) {
		Nd[i] = 1;
	}
	//GPU da d_Md, d_Nd, d_Pd de�i�kenleri i�in alan tahsisi//
	hipMalloc((void**)&d_Md, Width * Width * sizeof(float));
	hipMalloc((void**)&d_Nd, Width * 1 * sizeof(float));
	hipMalloc((void**)&d_Pd, Width * 1 * sizeof(float));
	//CPU daki verilerin GPU belle�ine at�lmas�//
	hipMemcpy(d_Md, Md, Width * Width * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Nd, Nd, Width * 1 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Pd, Pd, Width * 1 * sizeof(float), hipMemcpyHostToDevice);
	//matrisi b�lece�imiz bloklar�n ve ko�acak threadlar�n setlenmesi//
	dim3 dimGrid(Width / BLOCKSIZE, Width / BLOCKSIZE);
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	//parametlerin fonksiyona verilmesi//
	carpim << <Width / BLOCKSIZE, BLOCKSIZE >> >(d_Md, d_Nd, d_Pd);
	//geri d�nen d_Pd sonuc de�erinin GPU belle�inden CPU belle�ine aktar�lmas�//
	hipMemcpy(Pd, d_Pd, Width * 1 * sizeof(float), hipMemcpyDeviceToHost);
	//Paralel olarak hesaplanan ilk 10 degerin ekrana bas�lmas�//
	printf("ASAGIDA PROGRAM CIKTISININ ILK 10 DEGERI VERILMISTIR\n");
	for (int i = 0; i < 10; i++)
	{
		printf("%d. deger --> %.3f\n", i + 1, Pd[i]);
	}
	end = clock();
	//paralel s�re hesaplanmas� ve ekrana bas�lmas�//
	printf("Paralel Hesaplama Suresi --> %f(sn)\n", (((float)end - (float)start) / 1000000.0F) * 1000);
	free(Md);
	free(Nd);
	free(Pd);
	hipFree(d_Md);
	hipFree(d_Nd);
	hipFree(d_Pd);
	return 0;
}